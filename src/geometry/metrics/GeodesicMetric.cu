#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <queue>
#include <vector>
#include <iostream>
#include <hip/hip_runtime.h>

#define MAX_ITER 100
#define INF_FLT 1e30f

int TPB_geodesic = 256;

// Kernel: assign each face to the nearest centroid using a precomputed d_distances array [K*N] 
__global__ void kMeansClusterAssignmentGeodesic(
    const float* d_distances,  // [K*N]
    int*         d_clust_assn, // [N]
    int N,
    int K
)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx >= N) return;

    float minDist = INF_FLT;
    int bestC = -1;
    for(int c=0; c<K; c++){
        float dist = d_distances[c*N + idx];
        if(dist < minDist){
            minDist = dist;
            bestC   = c;
        }
    }
    d_clust_assn[idx] = bestC;
}

// Kernel: sum the baricenters of faces belonging to each cluster
__global__ void kMeansCentroidSumGeodesic(
    const float* d_faceBaricenter, // [N*dim]
    const int*   d_clust_assn,     // [N]
    float*       d_centroid_sums,  // [K*dim]
    int*         d_clust_sizes,    // [K]
    int N,
    int K,
    int dim
)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx >= N) return;

    int c = d_clust_assn[idx];
    // Accumulate
    for(int d=0; d<dim; d++){
        atomicAdd(&d_centroid_sums[c*dim + d], d_faceBaricenter[idx*dim + d]);
    }
    atomicAdd(&d_clust_sizes[c], 1);
}

// Kernel: update the centroid => sum / size
__global__ void kMeansCentroidUpdateGeodesic(
    float* d_centroids, 
    const float* d_centroid_sums,
    const int*   d_clust_sizes,
    int K,
    int dim
)
{
    int c = blockIdx.x*blockDim.x + threadIdx.x;
    if(c >= K) return;

    int size = d_clust_sizes[c];
    if(size > 0){
        for(int d=0; d<dim; d++){
            d_centroids[c*dim + d] = d_centroid_sums[c*dim + d] / (float)size;
        }
    } else {
        for(int d=0; d<dim; d++){
            d_centroids[c*dim + d] = 0.f;
        }
    }
}

// CPU Dijkstra to compute geodesic distances from a single "startFace"
static void dijkstraCPU(
    int startFace,
    int N,
    const std::vector<std::vector<int>>& adjacency,
    const std::vector<float>& faceBaricenter, // [N*dim]
    int dim,
    std::vector<float>& outDist              // [N]
)
{
    outDist.assign(N, INF_FLT);
    std::vector<bool> visited(N,false);

    using PQItem = std::pair<float,int>;
    std::priority_queue<PQItem, std::vector<PQItem>, std::greater<PQItem>> pq;

    outDist[startFace] = 0.f;
    pq.push({0.f, startFace});

    while(!pq.empty()){
        auto tmp    = pq.top();
        float dist  = tmp.first;
        int faceId  = tmp.second;

        pq.pop();
        if(visited[faceId]) continue;
        visited[faceId] = true;

        // For each neighbor
        for(int neigh : adjacency[faceId]){
            float w=0.f;
            for(int d=0; d<dim; d++){
                float diff = faceBaricenter[faceId*dim + d]
                           - faceBaricenter[neigh*dim + d];
                w += diff*diff;
            }
            w = sqrtf(w);

            float nd = dist + w;
            if(nd < outDist[neigh]){
                outDist[neigh] = nd;
                pq.push({nd, neigh});
            }
        }
    }
}

__global__ void findClosestFaceKernel(
    const float* d_faceBaricenter, // Array of face barycenters [N * dim]
    int N,                         // Total number of faces
    int dim,                       // Dimension (e.g. 3 for 3D)
    const float* d_centroid,       // Pointer to centroid coordinates (array of length dim)
    float* d_blockMinDistances,    // Output: minimum squared distance per block
    int* d_blockMinIndices         // Output: corresponding face index per block
)
{
    // Allocate shared memory: first blockDim.x floats for distances,
    // then blockDim.x ints for indices.
    extern __shared__ char sharedMem[];
    float* s_dist = reinterpret_cast<float*>(sharedMem);
    int* s_idx = reinterpret_cast<int*>(sharedMem + blockDim.x * sizeof(float));

    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + tid;

    // Initialize best distance to a large value.
    float bestDist = FLT_MAX;
    int bestIdx = -1;

    if (idx < N) {
        float dist = 0.0f;
        // Compute squared Euclidean distance between the face barycenter and the centroid.
        for (int d = 0; d < dim; d++) {
            float diff = d_faceBaricenter[idx * dim + d] - d_centroid[d];
            dist += diff * diff;
        }
        bestDist = dist;
        bestIdx = idx;
    }

    s_dist[tid] = bestDist;
    s_idx[tid] = bestIdx;
    __syncthreads();

    // Reduction in shared memory.
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            if (s_dist[tid + s] < s_dist[tid]) {
                s_dist[tid] = s_dist[tid + s];
                s_idx[tid] = s_idx[tid + s];
            }
        }
        __syncthreads();
    }

    // Write the block's result to global memory.
    if (tid == 0) {
        d_blockMinDistances[blockIdx.x] = s_dist[0];
        d_blockMinIndices[blockIdx.x] = s_idx[0];
    }
}

// Host function to find the closest face index for a given centroid using the GPU.
int findClosestFaceGPU(const float* d_faceBaricenter, int N, int dim, const float* d_centroid) {
    int numBlocks = (N + TPB_geodesic - 1) / TPB_geodesic;

    // Allocate memory for block-level results on the device.
    float* d_blockMinDistances;
    int* d_blockMinIndices;
    hipMalloc(&d_blockMinDistances, numBlocks * sizeof(float));
    hipMalloc(&d_blockMinIndices, numBlocks * sizeof(int));

    // Calculate shared memory size: each block uses TPB_geodesic * (sizeof(float) + sizeof(int)).
    size_t sharedMemSize = TPB_geodesic * (sizeof(float) + sizeof(int));

    // Launch the kernel.
    findClosestFaceKernel<<<numBlocks, TPB_geodesic, sharedMemSize>>>(
        d_faceBaricenter,
        N,
        dim,
        d_centroid,
        d_blockMinDistances,
        d_blockMinIndices
    );
    hipDeviceSynchronize();

    // Copy the block-level results back to host.
    std::vector<float> h_blockMinDistances(numBlocks);
    std::vector<int> h_blockMinIndices(numBlocks);
    hipMemcpy(h_blockMinDistances.data(), d_blockMinDistances, numBlocks * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_blockMinIndices.data(), d_blockMinIndices, numBlocks * sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory for block results.
    hipFree(d_blockMinDistances);
    hipFree(d_blockMinIndices);

    // Final reduction on host.
    float bestDist = FLT_MAX;
    int bestIdx = -1;
    for (int i = 0; i < numBlocks; i++) {
        if (h_blockMinDistances[i] < bestDist) {
            bestDist = h_blockMinDistances[i];
            bestIdx = h_blockMinIndices[i];
        }
    }
    return bestIdx;
}

static void setupGeodesicDistances(
    float* d_distances,                   // [K*N] device memory
    std::vector<float>& outDistancesHost, // [K*N], reused each iteration
    const float* d_faceBaricenter,        // device pointer to face barycenters
    const std::vector<float>& h_faceBaricenter, // host copy of face barycenters [N*dim]
    const std::vector<std::vector<int>>& adjacency,
    int N,
    int K,
    int dim,
    const float* d_centroids              // device pointer to centroids [K*dim]
)
{
    for (int c = 0; c < K; c++){
        // For the current centroid, compute the closest face index using the GPU.
        // d_centroids + c*dim points to the centroid 'c'.
        int startFace = findClosestFaceGPU(d_faceBaricenter, N, dim, d_centroids + c * dim);

        // Compute geodesic distances from the chosen startFace using the CPU Dijkstra.
        std::vector<float> distC(N);
        dijkstraCPU(startFace, N, adjacency, h_faceBaricenter, dim, distC);

        // Store the distances in the temporary host array.
        for (int f = 0; f < N; f++){
            outDistancesHost[c * N + f] = distC[f];
        }
    }

    // Copy the computed distances from host to device.
    hipMemcpy(d_distances, outDistancesHost.data(), K * N * sizeof(float), hipMemcpyHostToDevice);
}


void kmeans_cuda_geodesic(
    int N,  // faces
    int K,  // centroids
    int dim,
    std::vector<float>& h_faceBaricenter, // [N*dim] in
    std::vector<float>& h_centroids,      // [K*dim] in/out
    std::vector<int>&   h_faceCluster,    // [N] out
    const std::vector<std::vector<int>>& adjacency,
    float threshold
)
{
    std::cout << "TPB_geodesic: " << TPB_geodesic << std::endl;
    // Alloc device memory
    float* d_faceBaricenter = nullptr;
    float* d_centroids      = nullptr;
    float* d_distances      = nullptr; // [K*N]
    int*   d_clust_assn     = nullptr; // [N]

    hipMalloc(&d_faceBaricenter, N*dim*sizeof(float));
    hipMalloc(&d_centroids,      K*dim*sizeof(float));
    hipMalloc(&d_distances,      K*N*sizeof(float));
    hipMalloc(&d_clust_assn,     N*sizeof(int));

    float* d_centroid_sums = nullptr;
    hipMalloc(&d_centroid_sums, K*dim*sizeof(float));
    int* d_clust_sizes = nullptr;
    hipMalloc(&d_clust_sizes,   K*sizeof(int));

    // Copy initial data => device
    hipMemcpy(d_faceBaricenter,
               h_faceBaricenter.data(),
               N*dim*sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(d_centroids,
               h_centroids.data(),
               K*dim*sizeof(float),
               hipMemcpyHostToDevice);
    hipMemset(d_clust_assn, 0, N*sizeof(int));

    // We'll keep oldAssignment to check convergence
    std::vector<int> oldAssignment(N, -1);

    // We'll also keep a temporary host array for d_distances
    // to store the CPU-based Dijkstra results. size = K*N
    std::vector<float> tempHostDistances(K*N, 0.f);

    bool converged = false;
    int iteration  = 0;

    while(!converged && iteration < MAX_ITER)
    {
        // "setup" => compute geodesic distances with Dijkstra + copy to device
        setupGeodesicDistances(
            d_distances,
            tempHostDistances,
            d_faceBaricenter,  // device pointer to face barycenters
            h_faceBaricenter,  // host copy of face barycenters
            adjacency,
            N,
            K,
            dim,
            d_centroids        // device pointer to centroids
        );

        // cluster assignment on GPU
        {
            dim3 blocks((N + TPB_geodesic -1)/TPB_geodesic);
            kMeansClusterAssignmentGeodesic<<<blocks, TPB_geodesic>>>(
                d_distances, 
                d_clust_assn,
                N,
                K
            );
            hipDeviceSynchronize();
        }

        // reset sums / sizes
        hipMemset(d_centroid_sums, 0, K*dim*sizeof(float));
        hipMemset(d_clust_sizes,   0, K*sizeof(int));

        // sum
        {
            dim3 blocks((N + TPB_geodesic -1)/TPB_geodesic);
            kMeansCentroidSumGeodesic<<<blocks, TPB_geodesic>>>(
                d_faceBaricenter,
                d_clust_assn,
                d_centroid_sums,
                d_clust_sizes,
                N,
                K,
                dim
            );
            hipDeviceSynchronize();
        }

        // update
        {
            dim3 blocksC((K + TPB_geodesic-1)/TPB_geodesic);
            kMeansCentroidUpdateGeodesic<<<blocksC, TPB_geodesic>>>(
                d_centroids,
                d_centroid_sums,
                d_clust_sizes,
                K,
                dim
            );
            hipDeviceSynchronize();
        }

        // check how many changed => convergence
        {
            std::vector<int> tempAssign(N);
            hipMemcpy(tempAssign.data(), d_clust_assn,
                       N*sizeof(int), hipMemcpyDeviceToHost);

            unsigned int changed = 0;
            for(int i=0; i<N; i++){
                if(oldAssignment[i] != tempAssign[i]){
                    changed++;
                }
            }
            oldAssignment  = tempAssign;
            h_faceCluster  = tempAssign; // final assignment in host

            float ratio = (float)changed / (float)N;
            if(ratio <= threshold){
                converged = true;
            }
        }

        iteration++;
    }

    // Copy centroids back to host
    hipMemcpy(h_centroids.data(), d_centroids,
               K*dim*sizeof(float), hipMemcpyDeviceToHost);

    // free device memory
    hipFree(d_faceBaricenter);
    hipFree(d_centroids);
    hipFree(d_distances);
    hipFree(d_clust_assn);
    hipFree(d_centroid_sums);
    hipFree(d_clust_sizes);

    std::cout << "[kmeans_cuda_geodesic] finished after "
              << iteration << " iterations."
              << (converged ? " (converged)\n" : " (maxIter)\n");
}
