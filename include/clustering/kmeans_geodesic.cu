#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <queue>
#include <vector>
#include <iostream>
#include <hip/hip_runtime.h>

#define TPB 256
#define MAX_ITER 100
#define INF_FLT 1e30f

// Kernel: assign each face to the nearest centroid using a precomputed d_distances array [K*N] 
__global__ void kMeansClusterAssignmentGeodesic(
    const float* d_distances,  // [K*N]
    int*         d_clust_assn, // [N]
    int N,
    int K
)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx >= N) return;

    float minDist = INF_FLT;
    int bestC = -1;
    for(int c=0; c<K; c++){
        float dist = d_distances[c*N + idx];
        if(dist < minDist){
            minDist = dist;
            bestC   = c;
        }
    }
    d_clust_assn[idx] = bestC;
}

// Kernel: sum the baricenters of faces belonging to each cluster
__global__ void kMeansCentroidSumGeodesic(
    const float* d_faceBaricenter, // [N*dim]
    const int*   d_clust_assn,     // [N]
    float*       d_centroid_sums,  // [K*dim]
    int*         d_clust_sizes,    // [K]
    int N,
    int K,
    int dim
)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx >= N) return;

    int c = d_clust_assn[idx];
    // Accumulate
    for(int d=0; d<dim; d++){
        atomicAdd(&d_centroid_sums[c*dim + d], d_faceBaricenter[idx*dim + d]);
    }
    atomicAdd(&d_clust_sizes[c], 1);
}

// Kernel: update the centroid => sum / size
__global__ void kMeansCentroidUpdateGeodesic(
    float* d_centroids, 
    const float* d_centroid_sums,
    const int*   d_clust_sizes,
    int K,
    int dim
)
{
    int c = blockIdx.x*blockDim.x + threadIdx.x;
    if(c >= K) return;

    int size = d_clust_sizes[c];
    if(size > 0){
        for(int d=0; d<dim; d++){
            d_centroids[c*dim + d] = d_centroid_sums[c*dim + d] / (float)size;
        }
    } else {
        for(int d=0; d<dim; d++){
            d_centroids[c*dim + d] = 0.f;
        }
    }
}

// CPU Dijkstra to compute geodesic distances from a single "startFace"
static void dijkstraCPU(
    int startFace,
    int N,
    const std::vector<std::vector<int>>& adjacency,
    const std::vector<float>& faceBaricenter, // [N*dim]
    int dim,
    std::vector<float>& outDist              // [N]
)
{
    outDist.assign(N, INF_FLT);
    std::vector<bool> visited(N,false);

    using PQItem = std::pair<float,int>;
    std::priority_queue<PQItem, std::vector<PQItem>, std::greater<PQItem>> pq;

    outDist[startFace] = 0.f;
    pq.push({0.f, startFace});

    while(!pq.empty()){
        auto tmp    = pq.top();
        float dist  = tmp.first;
        int faceId  = tmp.second;

        pq.pop();
        if(visited[faceId]) continue;
        visited[faceId] = true;

        // For each neighbor
        for(int neigh : adjacency[faceId]){
            float w=0.f;
            for(int d=0; d<dim; d++){
                float diff = faceBaricenter[faceId*dim + d]
                           - faceBaricenter[neigh*dim + d];
                w += diff*diff;
            }
            w = sqrtf(w);

            float nd = dist + w;
            if(nd < outDist[neigh]){
                outDist[neigh] = nd;
                pq.push({nd, neigh});
            }
        }
    }
}

static void setupGeodesicDistances(
    float* d_distances,                   // [K*N] device memory
    std::vector<float>& outDistancesHost, // [K*N], reused each iteration
    const std::vector<float>& faceBaricenter, // [N*dim]
    const std::vector<std::vector<int>>& adjacency,
    int N,
    int K,
    int dim
)
{

    for(int c=0; c<K; c++){
        int startFace = c; // or some "closestFaceId[c]"
        std::vector<float> distC(N);
        dijkstraCPU(startFace, N, adjacency, faceBaricenter, dim, distC);
        for(int f=0; f<N; f++){
            outDistancesHost[c*N + f] = distC[f];
        }
    }

    // outDistancesHost => d_distances (device)
    hipMemcpy(d_distances,
               outDistancesHost.data(),
               K*N*sizeof(float),
               hipMemcpyHostToDevice);
}


void kmeans_cuda_geodesic(
    int N,  // faces
    int K,  // centroids
    int dim,
    std::vector<float>& h_faceBaricenter, // [N*dim] in
    std::vector<float>& h_centroids,      // [K*dim] in/out
    std::vector<int>&   h_faceCluster,    // [N] out
    const std::vector<std::vector<int>>& adjacency,
    float threshold
)
{
    // Alloc device memory
    float* d_faceBaricenter = nullptr;
    float* d_centroids      = nullptr;
    float* d_distances      = nullptr; // [K*N]
    int*   d_clust_assn     = nullptr; // [N]

    hipMalloc(&d_faceBaricenter, N*dim*sizeof(float));
    hipMalloc(&d_centroids,      K*dim*sizeof(float));
    hipMalloc(&d_distances,      K*N*sizeof(float));
    hipMalloc(&d_clust_assn,     N*sizeof(int));

    float* d_centroid_sums = nullptr;
    hipMalloc(&d_centroid_sums, K*dim*sizeof(float));
    int* d_clust_sizes = nullptr;
    hipMalloc(&d_clust_sizes,   K*sizeof(int));

    // Copy initial data => device
    hipMemcpy(d_faceBaricenter,
               h_faceBaricenter.data(),
               N*dim*sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(d_centroids,
               h_centroids.data(),
               K*dim*sizeof(float),
               hipMemcpyHostToDevice);
    hipMemset(d_clust_assn, 0, N*sizeof(int));

    // We'll keep oldAssignment to check convergence
    std::vector<int> oldAssignment(N, -1);

    // We'll also keep a temporary host array for d_distances
    // to store the CPU-based Dijkstra results. size = K*N
    std::vector<float> tempHostDistances(K*N, 0.f);

    bool converged = false;
    int iteration  = 0;

    while(!converged && iteration < MAX_ITER)
    {
        // "setup" => compute geodesic distances with Dijkstra + copy to device
        setupGeodesicDistances(
            d_distances,
            tempHostDistances,
            h_faceBaricenter,
            adjacency,
            N,
            K,
            dim
        );

        // cluster assignment on GPU
        {
            dim3 blocks((N + TPB -1)/TPB);
            kMeansClusterAssignmentGeodesic<<<blocks, TPB>>>(
                d_distances, 
                d_clust_assn,
                N,
                K
            );
            hipDeviceSynchronize();
        }

        // reset sums / sizes
        hipMemset(d_centroid_sums, 0, K*dim*sizeof(float));
        hipMemset(d_clust_sizes,   0, K*sizeof(int));

        // sum
        {
            dim3 blocks((N + TPB -1)/TPB);
            kMeansCentroidSumGeodesic<<<blocks, TPB>>>(
                d_faceBaricenter,
                d_clust_assn,
                d_centroid_sums,
                d_clust_sizes,
                N,
                K,
                dim
            );
            hipDeviceSynchronize();
        }

        // update
        {
            dim3 blocksC((K + TPB-1)/TPB);
            kMeansCentroidUpdateGeodesic<<<blocksC, TPB>>>(
                d_centroids,
                d_centroid_sums,
                d_clust_sizes,
                K,
                dim
            );
            hipDeviceSynchronize();
        }

        // check how many changed => convergence
        {
            std::vector<int> tempAssign(N);
            hipMemcpy(tempAssign.data(), d_clust_assn,
                       N*sizeof(int), hipMemcpyDeviceToHost);

            unsigned int changed = 0;
            for(int i=0; i<N; i++){
                if(oldAssignment[i] != tempAssign[i]){
                    changed++;
                }
            }
            oldAssignment  = tempAssign;
            h_faceCluster  = tempAssign; // final assignment in host

            float ratio = (float)changed / (float)N;
            if(ratio <= threshold){
                converged = true;
            }
        }

        iteration++;
    }

    // Copy centroids back to host
    hipMemcpy(h_centroids.data(), d_centroids,
               K*dim*sizeof(float), hipMemcpyDeviceToHost);

    // free device memory
    hipFree(d_faceBaricenter);
    hipFree(d_centroids);
    hipFree(d_distances);
    hipFree(d_clust_assn);
    hipFree(d_centroid_sums);
    hipFree(d_clust_sizes);

    std::cout << "[kmeans_cuda_geodesic] finished after "
              << iteration << " iterations."
              << (converged ? " (converged)\n" : " (maxIter)\n");
}
