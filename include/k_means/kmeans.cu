#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <hip/hip_runtime.h>

#define TPB 256
#define MAX_ITER 100

// Compute the Euclidean distance between two points in n-dimensional space
__device__ float distance(const float *x1, const float *x2, int dim) {
    float sum = 0.0f;
    for (int i = 0; i < dim; ++i) {
        float diff = x2[i] - x1[i];
        sum += diff * diff;
    }
    return sqrtf(sum);
}

// Assign each point to the closest centroid
__global__ void kMeansClusterAssignment(const float *d_datapoints, int *d_clust_assn, const float *d_centroids, int N, int K, int dim) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= N) return;

    float min_dist = INFINITY;
    int closest_centroid = -1;

    for (int c = 0; c < K; ++c) {
        float dist = distance(&d_datapoints[idx * dim], &d_centroids[c * dim], dim);
        if (dist < min_dist) {
            min_dist = dist;
            closest_centroid = c;
        }
    }

    d_clust_assn[idx] = closest_centroid;
}

// Compute the sums of datapoints assigned to each cluster
__global__ void kMeansCentroidSum(const float *d_datapoints, const int *d_clust_assn, float *d_centroid_sums, int *d_clust_sizes, int N, int K, int dim) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= N) return;

    int cluster_id = d_clust_assn[idx];

    for (int d = 0; d < dim; ++d) {
        atomicAdd(&d_centroid_sums[cluster_id * dim + d], d_datapoints[idx * dim + d]);
    }
    atomicAdd(&d_clust_sizes[cluster_id], 1);
}

// Update the centroids
__global__ void kMeansCentroidUpdate(float *d_centroids, const float *d_centroid_sums, const int *d_clust_sizes, int K, int dim) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= K) return;

    for (int d = 0; d < dim; ++d) {
        int sum_idx = idx * dim + d;
        if (d_clust_sizes[idx] > 0) {
            d_centroids[sum_idx] = d_centroid_sums[sum_idx] / d_clust_sizes[idx];
        } else {
            d_centroids[sum_idx] = 0.0f;
        }
    }
}

void kmeans_cuda(int K, int dim, int numPoints, float *points, float *centroids, int *clust_assn, float threshold) {
    float *d_datapoints = nullptr, *d_centroids = nullptr, *d_centroid_sums = nullptr;
    int *d_clust_assn = nullptr, *d_clust_sizes = nullptr;

    hipMalloc(&d_datapoints, numPoints * dim * sizeof(float));
    hipMalloc(&d_centroids, K * dim * sizeof(float));
    hipMalloc(&d_centroid_sums, K * dim * sizeof(float));
    hipMalloc(&d_clust_assn, numPoints * sizeof(int));
    hipMalloc(&d_clust_sizes, K * sizeof(int));

    hipMemcpy(d_datapoints, points, numPoints * dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_centroids, centroids, K * dim * sizeof(float), hipMemcpyHostToDevice);

    dim3 blocks_points((numPoints + TPB - 1) / TPB);
    dim3 blocks_clusters((K + TPB - 1) / TPB);

    for (int iter = 0; iter < MAX_ITER; ++iter) {
        kMeansClusterAssignment<<<blocks_points, TPB>>>(d_datapoints, d_clust_assn, d_centroids, numPoints, K, dim);

        hipMemset(d_centroid_sums, 0, K * dim * sizeof(float));
        hipMemset(d_clust_sizes, 0, K * sizeof(int));

        kMeansCentroidSum<<<blocks_points, TPB>>>(d_datapoints, d_clust_assn, d_centroid_sums, d_clust_sizes, numPoints, K, dim);
        kMeansCentroidUpdate<<<blocks_clusters, TPB>>>(d_centroids, d_centroid_sums, d_clust_sizes, K, dim);
    }

    hipMemcpy(centroids, d_centroids, K * dim * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(clust_assn, d_clust_assn, numPoints * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_datapoints);
    hipFree(d_centroids);
    hipFree(d_centroid_sums);
    hipFree(d_clust_assn);
    hipFree(d_clust_sizes);
}
